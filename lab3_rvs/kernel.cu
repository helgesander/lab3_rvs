﻿#include <hip/hip_runtime.h>
#include "wb.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/functional.h>

int main(int argc, char* argv[]) {
	wbArg_t args;
	float* hostInput1;
	float* hostInput2;
	int inputLength;

	args = wbArg_read(argc, argv); // Чтение входных аргументов 
	wbTime_start(Generic, "Import data to host");
	hostInput1 = (float*)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float*)wbImport(wbArg_getInputFile(args, 1), &inputLength);
	//wbTime_stop(Generic, "Importing data to host"); --> вылазит непонятная ошибка 


	// Объявление и выделение памяти под выходные данные
	float* hostOutput = (float*)malloc(sizeof(float) * inputLength);
	//@@ Место для вставки кода
	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	// Объявление и выделение памяти под входные и выходные данные  на устройства через thrust
	wbTime_start(GPU, "Doing GPU memory allocation");
	thrust::device_vector<float> input1(inputLength);
	thrust::device_vector<float> input2(inputLength);
	thrust::device_vector<float> output(inputLength);
	wbTime_start(Copy, "Copying data to the GPU");
	//@@ Место для вставки кода
	thrust::copy(hostInput1, hostInput1 + inputLength, input1.begin());
	thrust::copy(hostInput2, hostInput2 + inputLength, input2.begin());
	wbTime_stop(Copy, "Copying data to the GPU");

	// Выполнение операции сложения векторов
	wbTime_start(Compute, "Doing the computation on the GPU");
	//@@ Место для вставки кода
	thrust::transform(input1.begin(), input1.end(), input2.begin(), output.begin(), thrust::plus<float>());
	wbTime_stop(Compute, "Doing the computation on the GPU");
	/////////////////////////////////////////////////////////

	// Копирование данных обратно на хост
	wbTime_start(Copy, "Copying data from the GPU");
	//@@ Место для вставки кода
	thrust::copy(output.begin(), output.end(), hostOutput);
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);
	return 0;
}